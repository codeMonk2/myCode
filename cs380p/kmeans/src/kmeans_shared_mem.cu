#include "hip/hip_runtime.h"
#include<stdio.h>
#include<limits>
#include "argparse.h"
#include "helpers.h"

__device__ double pointDistanceShMem(unsigned int dims, double *point, double *centroid)
{
    double sum = 0.0;

    for(unsigned int i = 0; i < dims; i++)
    {
        sum += pow((point[i]-centroid[i]), 2.0);
    }

    return sqrt(sum);
}

__global__ void cudaNewCentroids3(struct options_t *d_opts,
                                  double *d_centroids,
                                  double *d_centroidSums,
                                  int    *d_ExceedThreshold,
                                  int    *d_numPointsAtCentroid)
{
    const int centroid = threadIdx.x;
    const int count = max(1, d_numPointsAtCentroid[centroid]);

    for (int j = 0; j < d_opts->nDims; j++)
    {
        if(((double)d_centroidSums[(centroid*d_opts->nDims)+j]/count) - (double)d_centroids[(centroid*d_opts->nDims)+j] > d_opts->threshold)
        {
			atomicAdd(d_ExceedThreshold, 1);
        }
        d_centroids[(centroid*d_opts->nDims)+j] = d_centroidSums[(centroid*d_opts->nDims)+j]/count;
        d_centroidSums[(centroid*d_opts->nDims)+j] = 0;
    }
    d_numPointsAtCentroid[centroid] = 0;
}

__global__ void cudaKmeans3(struct options_t *d_opts,
                            double *d_inputData,
                            double *d_centroids,
                            double *d_centroidSums,
                            int    *d_numPointsAtCentroid,
                            int    *d_pointCentroidMap,
                            double maxDistance)
{
	extern __shared__ double sharedCentroids[];

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    double distance = 0.0, bestDistance = maxDistance;
    int bestCentroid = 0;

    if (threadIdx.x < d_opts->nCentroids) 
    {
        for(int i = 0; i < d_opts->nDims; i++)
            sharedCentroids[(threadIdx.x*d_opts->nDims)+i] = d_centroids[(threadIdx.x*d_opts->nDims)+i];
    }
    __syncthreads();

    for(int j = 0; j < d_opts->nCentroids; j++)
    {
        distance = pointDistanceShMem(d_opts->nDims, &d_inputData[(index*d_opts->nDims)], &sharedCentroids[(j*d_opts->nDims)]);
        if(distance < bestDistance)
        {
            bestDistance = distance;
            bestCentroid = j;
        }
    }

    for (int j = 0; j < d_opts->nDims; j++)
    {
        atomicAdd(&d_centroidSums[(bestCentroid*d_opts->nDims)+j], d_inputData[(index*d_opts->nDims)+j]);
    }
    atomicAdd(&d_numPointsAtCentroid[bestCentroid], 1);
    d_pointCentroidMap[index]=bestCentroid;

    __syncthreads();
}

void runKmeansSharedMem(struct options_t *opts, double *inputData)
{
    double *d_inputData, *d_centroids, *d_centroidSums, *centroids;
    int *d_ExceedThreshold, *d_numPointsAtCentroid, *d_pointCentroidMap, *pointCentroidMap, n = 0, threads = 0, blocks = 0, exceedThreshold = 0;
    struct options_t *d_opts;
    hipDeviceProp_t prop;
    hipEvent_t start, stop;
    float delta_ms=0.0;

    hipGetDeviceProperties(&prop, 0);

    threads = prop.maxThreadsPerBlock;
    blocks = (opts->nVals+threads-1) / threads;

    initCentroids(opts, inputData, &centroids);
    pointCentroidMap = (int *)malloc((opts->nVals * sizeof(int)));
    memset(pointCentroidMap, 0, (opts->nVals * sizeof(int)));

    hipMalloc(&d_opts, sizeof(struct options_t));
    hipMemcpy(d_opts, opts, sizeof(struct options_t), hipMemcpyHostToDevice);

    hipMalloc(&d_inputData, ((opts->nVals * opts->nDims) * sizeof(double)));
    hipMemcpy(d_inputData, inputData, ((opts->nVals * opts->nDims) * sizeof(double)), hipMemcpyHostToDevice);

    hipMalloc(&d_centroids, ((opts->nCentroids * opts->nDims) * sizeof(double)));
    hipMemcpy(d_centroids, centroids, ((opts->nCentroids * opts->nDims) * sizeof(double)), hipMemcpyHostToDevice);

    hipMalloc(&d_centroidSums, ((opts->nCentroids * opts->nDims) * sizeof(double)));
    hipMalloc(&d_numPointsAtCentroid, (opts->nCentroids*sizeof(int)));
    hipMalloc(&d_pointCentroidMap, (opts->nVals * sizeof(int)));
    hipMalloc(&d_ExceedThreshold, sizeof(int));

    hipMemset(d_centroidSums, 0, ((opts->nCentroids * opts->nDims) * sizeof(double)));
    hipMemset(d_numPointsAtCentroid, 0, (opts->nCentroids*sizeof(int)));
    hipMemset(d_pointCentroidMap, 0, (opts->nVals * sizeof(int)));
	hipMemset(d_ExceedThreshold, 0, sizeof(int));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    do {
		hipMemset(d_ExceedThreshold, 0, sizeof(int));

        cudaKmeans3<<<blocks,threads, ((opts->nCentroids * opts->nDims) * sizeof(double))>>>(d_opts, d_inputData, d_centroids,
                                        d_centroidSums, d_numPointsAtCentroid,
                                        d_pointCentroidMap,
                                        std::numeric_limits<double>::max());

        cudaNewCentroids3<<<1,opts->nCentroids>>>(d_opts, d_centroids,
                            d_centroidSums, d_ExceedThreshold, d_numPointsAtCentroid);

		hipMemcpy(&exceedThreshold, d_ExceedThreshold, sizeof(int), hipMemcpyDeviceToHost);

        n++;
    } while (n < opts->nIters && exceedThreshold != 0);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&delta_ms, start, stop);

    hipMemcpy(centroids, d_centroids, ((opts->nCentroids * opts->nDims) * sizeof(double)), hipMemcpyDeviceToHost);
    hipMemcpy(pointCentroidMap, d_pointCentroidMap, (opts->nVals * sizeof(int)), hipMemcpyDeviceToHost);

    printf("%d,%lf\n", n, delta_ms/n);

    if (opts->bPrintCentroids)
    {
        printCentroids(opts, centroids);
    }
    else
    {
        printf("clusters:");
        for (int p=0; p < opts->nVals; p++)
            printf(" %d", pointCentroidMap[p]);
    }

    hipFree(d_opts);
    hipFree(d_inputData);
    hipFree(d_centroids);
    hipFree(d_centroidSums);
    hipFree(d_numPointsAtCentroid);
    hipFree(d_pointCentroidMap);
	free(pointCentroidMap);
    clearCentroids(&centroids);
}
