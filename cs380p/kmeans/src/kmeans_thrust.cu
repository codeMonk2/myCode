#include "hip/hip_runtime.h"
#include<stdio.h>
#include<limits>
#include "argparse.h"
#include "helpers.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

//{
    //for(unsigned int i = 0; i < opts->nVals; i++)
    //{
    //    //thrust::device_vector<double>::iterator inputIt = d_inputData.begin()+(i * opts->nDims);
    //    bestDistance = std::numeric_limits<double>::max();
    //    bestCentroid = 0;
    //    for(int j = 0; j < opts->nCentroids; j++)
    //    {
    //        //auto centroidIt = d_centroids.begin()+((opts->nDims)*j);
    
    //        distance = sqrt(thrust::transform_reduce(
    //                        thrust::device,
    //                        thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin()+(i * opts->nDims), (d_centroids.begin()+((opts->nDims)*j)))),
    //                        thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin()+(i * opts->nDims)+(opts->nDims), (d_centroids.begin()+((opts->nDims)*j)+(opts->nDims)))),
    //                        squaredDistance(),
    //                        0.0,
    //                        reduceDistance()));
    
    //        if(distance < bestDistance)
    //        {
    //            bestDistance = distance;
    //            bestCentroid = j;
    //        }
    //    }
    //    for(int cnt = 0; cnt < opts->nDims; cnt++)
    //        d_pointCentroidKeys[(i*opts->nDims)+cnt] = (bestCentroid*opts->nDims)+cnt;
    //}
//}

struct BestCentroidForInput {
    struct options_t *dOpts;
    double *dInput;
    double *dCentroids;
    int    *dPointCentroidKeys;
    double dMaxValue;
    explicit BestCentroidForInput(struct options_t *_dopts,
                          double *_dInput,
                          double *_dCentroids,
                          int    *_dPointCentroidKeys,
                          double _dMaxValue):
                          dOpts(_dopts),
                          dInput(_dInput),
                          dCentroids(_dCentroids),
                          dPointCentroidKeys(_dPointCentroidKeys),
                          dMaxValue(_dMaxValue) {}
    __host__ __device__ int operator()(int index)
    {
        double distance = 0.0, bestDistance = dMaxValue;
        int bestCentroid = 0;
	    double sum = 0.0;

        for(int j = 0; j < dOpts->nCentroids; j++)
        {
			sum = 0.0;
		    for(unsigned int i = 0; i < dOpts->nDims; i++)
		    {
		    	sum += pow((dInput[index+i]-dCentroids[(j*dOpts->nDims)+i]), 2.0);
		    }

		    distance = sqrt(sum);
            if(distance < bestDistance)
            {
                bestDistance = distance;
                bestCentroid = j;
            }
        }

        for(int cnt = 0; cnt < dOpts->nDims; cnt++)
            dPointCentroidKeys[index+cnt] = (bestCentroid*dOpts->nDims)+cnt;

		return bestCentroid;
    }
};

struct TestThreshold {
    const double a;
    TestThreshold(double _a):a(_a) {}
    __host__ __device__ double operator()(const thrust::tuple<double,double>&arg) {
        return (thrust::get<0>(arg) - thrust::get<1>(arg)) > a ? 1.0:0.0;  
    }
};

struct reduceAndCheckWithinThreshold {
    __host__ __device__ int operator()(double x, double y) {
        return (int)x+(int)y;
    }
};

struct meanOfSums {
    __host__ __device__ double operator()(double &x, double &y) {
        return (x/y); 
    }
};

struct squaredDistance {
    __host__ __device__ double operator()(const thrust::tuple<double,double>&arg) {
        return pow((thrust::get<0>(arg) - thrust::get<1>(arg)), 2.0);
    }
};

struct reduceDistance {
    __host__ __device__ double operator()(double x, double y) {
        return x+y;
    }
};

struct sumCentroidsAndCount
{
    __host__ __device__
    thrust::tuple<double,double> operator ()(const thrust::tuple<double,double>& lhs, const thrust::tuple<double,double>& rhs)
    {
        return thrust::make_tuple(
            thrust::get<0>(lhs) + thrust::get<0>(rhs),
            thrust::get<1>(lhs) + thrust::get<1>(rhs)
        );
    }
};

void runKmeansThrustCuda(struct options_t *opts, double *inputData)
{
    double *centroids;
    int n = 0;
    hipEvent_t start, stop;
    float delta_ms=0.0;
    struct options_t *d_opts;
    int *pointCentroidMap;

    initCentroids(opts, inputData, &centroids);

    hipMalloc(&d_opts, sizeof(struct options_t));
    hipMemcpy(d_opts, opts, sizeof(struct options_t), hipMemcpyHostToDevice);

    pointCentroidMap = (int *)malloc((opts->nVals * sizeof(int)));
    memset(pointCentroidMap, 0, (opts->nVals * sizeof(int)));

    thrust::device_vector<double>d_inputData(inputData, inputData+(opts->nVals*opts->nDims));
    thrust::device_vector<double>d_inputData1(inputData, inputData+(opts->nVals*opts->nDims));

    thrust::device_vector<double>d_centroids(centroids, centroids+(opts->nCentroids*opts->nDims));
    thrust::device_vector<int>d_pointCentroidKeys(opts->nVals*opts->nDims);

    thrust::device_vector<int>d_inputIndex(opts->nVals);
    thrust::sequence(thrust::device, d_inputIndex.begin(), d_inputIndex.end(), 0, opts->nDims);

    thrust::device_vector<double> d_centroidPointCount(opts->nVals*opts->nDims);
    thrust::device_vector<int> d_pointCentroidMap(opts->nVals);

    int withinThreshold = 0;

    thrust::equal_to<double> binary_pred;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    do {
        withinThreshold=0;
        d_inputData = d_inputData1;

        /* 
         * Transform the input by taking input index in strides of nDims
         * and computing the best centroid for each point in the Functor.
         * The index here acts as the index in a CUDA kernel.
         * The returned vector contains the best centroid selected for each input
         */
        thrust::transform(thrust::device, d_inputIndex.begin(), d_inputIndex.end(),
                          d_pointCentroidMap.begin(), 
                          BestCentroidForInput(d_opts,
                                       thrust::raw_pointer_cast(d_inputData.data()),
                                       thrust::raw_pointer_cast(d_centroids.data()),
                                       thrust::raw_pointer_cast(d_pointCentroidKeys.data()),
                                       std::numeric_limits<double>::max()));
        /*
         * The point centroid keys vector contains the best centroid + nDims index on all the dimensions
         * of each input. Sort by key sorts this set of keys to a set which consecutive indices contain the
         * columns of each input assigned to a certain centroid. The input is sorted according to this key array.
         */
        thrust::stable_sort_by_key(thrust::device, d_pointCentroidKeys.begin(), d_pointCentroidKeys.end(), d_inputData.begin());
        
        /*
         * Here the centroids assigned to a particular centroid are summed up and number of points assigned to
         * each centroid are counted.
         */
        thrust::reduce_by_key(d_pointCentroidKeys.begin(), d_pointCentroidKeys.end(),
            thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin(), thrust::make_constant_iterator(1.0))),
            d_pointCentroidKeys.begin(),
            thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin(), d_centroidPointCount.begin())),
            binary_pred,
            sumCentroidsAndCount());
        /*
         * We get mean value of the current assignments. This is the new centroid.
         */
        thrust::transform(thrust::device, d_inputData.begin(), d_inputData.begin()+(opts->nCentroids*opts->nDims),
                          d_centroidPointCount.begin(),  d_inputData.begin(), meanOfSums());

        /*
         * Check whether the old and new centroids are within a threshold. This is done by comparing the old and new
         * centroids and saving a 0 or 1. This is then reduced to check the sum and see if it is > 0 or = 0.
         */
        withinThreshold = thrust::transform_reduce(thrust::device, 
                                 thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin(), d_centroids.begin())), 
                                 thrust::make_zip_iterator(thrust::make_tuple(d_inputData.begin()+(opts->nCentroids*opts->nDims), d_centroids.begin()+(opts->nCentroids*opts->nDims))),
                                 TestThreshold(opts->threshold),
                                 0.0,
                                 reduceAndCheckWithinThreshold());

        thrust::copy(d_inputData.begin(), d_inputData.begin()+(opts->nCentroids*opts->nDims), d_centroids.begin());

        n++;
    } while(n < opts->nIters && withinThreshold != 0);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&delta_ms, start, stop);

    printf("%d,%lf\n", n, delta_ms/n);

    hipMemcpy(centroids, thrust::raw_pointer_cast(d_centroids.data()), ((opts->nCentroids*opts->nDims) * sizeof(double)), hipMemcpyDeviceToHost); 
    hipMemcpy(pointCentroidMap, thrust::raw_pointer_cast(d_pointCentroidMap.data()), (opts->nVals * sizeof(int)), hipMemcpyDeviceToHost);

    if (opts->bPrintCentroids)
    {
        printCentroids(opts, centroids);
    }
    else
    {
        printf("clusters:");
        for (int p=0; p < opts->nVals; p++)
            printf(" %d", (int)pointCentroidMap[p]);
    }

    hipFree(d_opts);
    free(pointCentroidMap);
    clearCentroids(&centroids);
}
